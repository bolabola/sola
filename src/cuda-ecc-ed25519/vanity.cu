#include "hip/hip_runtime.h"
#include <vector>
#include <random>
#include <chrono>

#include <iostream>
#include <ctime>

#include <assert.h>
#include <inttypes.h>
#include <pthread.h>
#include <stdio.h>

#include "hiprand/hiprand_kernel.h"
#include "ed25519.h"
#include "fixedint.h"
#include "gpu_common.h"
#include "gpu_ctx.h"

#include "keypair.cu"
#include "sc.cu"
#include "fe.cu"
#include "ge.cu"
#include "sha512.cu"
#include "../config.h"

/* -- Types ----------------------------------------------------------------- */

typedef struct {
	// CUDA Random States.
	hiprandState*    states[8];
} config;

/* -- Prototypes, Because C++ ----------------------------------------------- */

void            vanity_setup(config& vanity);
void            vanity_run(config& vanity);
void __global__ vanity_init(unsigned long long int* seed, hiprandState* state);
void __global__ vanity_scan(hiprandState* state, int* keys_found, int* gpu, int* execution_count);
bool __device__ b58enc(char* b58, size_t* b58sz, uint8_t* data, size_t binsz);

/* -- Entry Point ----------------------------------------------------------- */

int main(int argc, char const* argv[]) {
	ed25519_set_verbose(true);

	config vanity;
	vanity_setup(vanity);
	vanity_run(vanity);
}

// SMITH
std::string getTimeStr(){
    std::time_t now = std::chrono::system_clock::to_time_t(std::chrono::system_clock::now());
    std::string s(30, '\0');
    std::strftime(&s[0], s.size(), "%Y-%m-%d %H:%M:%S", std::localtime(&now));
    return s;
}

// SMITH - safe? who knows
unsigned long long int makeSeed() {
    unsigned long long int seed = 0;
    char *pseed = (char *)&seed;

    std::random_device rd;

    for(unsigned int b=0; b<sizeof(seed); b++) {
      auto r = rd();
      char *entropy = (char *)&r;
      pseed[b] = entropy[0];
    }

    return seed;
}

/* -- Vanity Step Functions ------------------------------------------------- */

void vanity_setup(config &vanity) {
	printf("GPU: Initializing Memory\n");
	int gpuCount = 0;
	hipGetDeviceCount(&gpuCount);

	// Create random states so kernels have access to random generators
	// while running in the GPU.
	for (int i = 0; i < gpuCount; ++i) {
		hipSetDevice(i);

		// Fetch Device Properties
		hipDeviceProp_t device;
		hipGetDeviceProperties(&device, i);

		// Calculate Occupancy
		int blockSize       = 0,
		    minGridSize     = 0,
		    maxActiveBlocks = 0;
		hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, vanity_scan, 0, 0);
		hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, vanity_scan, blockSize, 0);

		// Output Device Details
		// 
		// Our kernels currently don't take advantage of data locality
		// or how warp execution works, so each thread can be thought
		// of as a totally independent thread of execution (bad). On
		// the bright side, this means we can really easily calculate
		// maximum occupancy for a GPU because we don't have to care
		// about building blocks well. Essentially we're trading away
		// GPU SIMD ability for standard parallelism, which CPUs are
		// better at and GPUs suck at.
		//
		// Next Weekend Project: ^ Fix this.
		printf("GPU: %d (%s <%d, %d, %d>) -- W: %d, P: %d, TPB: %d, MTD: (%dx, %dy, %dz), MGS: (%dx, %dy, %dz)\n",
			i,
			device.name,
			blockSize,
			minGridSize,
			maxActiveBlocks,
			device.warpSize,
			device.multiProcessorCount,
		       	device.maxThreadsPerBlock,
			device.maxThreadsDim[0],
			device.maxThreadsDim[1],
			device.maxThreadsDim[2],
			device.maxGridSize[0],
			device.maxGridSize[1],
			device.maxGridSize[2]
		);

                // the random number seed is uniquely generated each time the program 
                // is run, from the operating system entropy

		unsigned long long int rseed = makeSeed();
		printf("Initialising from entropy: %llu\n",rseed);

		unsigned long long int* dev_rseed;
	        hipMalloc((void**)&dev_rseed, sizeof(unsigned long long int));		
                hipMemcpy( dev_rseed, &rseed, sizeof(unsigned long long int), hipMemcpyHostToDevice ); 

		hipMalloc((void **)&(vanity.states[i]), maxActiveBlocks * blockSize * sizeof(hiprandState));
		vanity_init<<<maxActiveBlocks, blockSize>>>(dev_rseed, vanity.states[i]);
	}

	printf("END: Initializing Memory\n");
}

void vanity_run(config &vanity) {
	int gpuCount = 0;
	hipGetDeviceCount(&gpuCount);

	unsigned long long int  executions_total = 0; 
	unsigned long long int  executions_this_iteration; 
	int  executions_this_gpu; 
        int* dev_executions_this_gpu[100];

        int  keys_found_total = 0;
        int  keys_found_this_iteration;
        int* dev_keys_found[100]; // not more than 100 GPUs ok!

	for (int i = 0; i < MAX_ITERATIONS; ++i) {
		auto start  = std::chrono::high_resolution_clock::now();

                executions_this_iteration=0;

		// Run on all GPUs
		for (int g = 0; g < gpuCount; ++g) {
			hipSetDevice(g);
			// Calculate Occupancy
			int blockSize       = 0,
			    minGridSize     = 0,
			    maxActiveBlocks = 0;
			hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, vanity_scan, 0, 0);
			hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, vanity_scan, blockSize, 0);

			int* dev_g;
	                hipMalloc((void**)&dev_g, sizeof(int));
                	hipMemcpy( dev_g, &g, sizeof(int), hipMemcpyHostToDevice ); 

	                hipMalloc((void**)&dev_keys_found[g], sizeof(int));		
	                hipMalloc((void**)&dev_executions_this_gpu[g], sizeof(int));		

			vanity_scan<<<maxActiveBlocks, blockSize>>>(vanity.states[g], dev_keys_found[g], dev_g, dev_executions_this_gpu[g]);

		}

		// Synchronize while we wait for kernels to complete. I do not
		// actually know if this will sync against all GPUs, it might
		// just sync with the last `i`, but they should all complete
		// roughly at the same time and worst case it will just stack
		// up kernels in the queue to run.
		hipDeviceSynchronize();
		auto finish = std::chrono::high_resolution_clock::now();

		for (int g = 0; g < gpuCount; ++g) {
                	hipMemcpy( &keys_found_this_iteration, dev_keys_found[g], sizeof(int), hipMemcpyDeviceToHost ); 
                	keys_found_total += keys_found_this_iteration; 
			//printf("GPU %d found %d keys\n",g,keys_found_this_iteration);

                	hipMemcpy( &executions_this_gpu, dev_executions_this_gpu[g], sizeof(int), hipMemcpyDeviceToHost ); 
                	executions_this_iteration += executions_this_gpu * ATTEMPTS_PER_EXECUTION; 
                	executions_total += executions_this_gpu * ATTEMPTS_PER_EXECUTION; 
                        //printf("GPU %d executions: %d\n",g,executions_this_gpu);
		}

		// Print out performance Summary
		std::chrono::duration<double> elapsed = finish - start;
		/*printf("%s Iteration %d Attempts: %llu in %f at %fcps - Total Attempts %llu - keys found %d\n",
			getTimeStr().c_str(),
			i+1,
			executions_this_iteration, //(8 * 8 * 256 * 100000),
			elapsed.count(),
			executions_this_iteration / elapsed.count(),
			executions_total,
			keys_found_total
		);*/

                if ( keys_found_total >= STOP_AFTER_KEYS_FOUND ) {
                	printf("Enough keys found, Done! \n");
		        exit(0);	
		}	
	}

	printf("Iterations complete, Done!\n");
}

/* -- CUDA Vanity Functions ------------------------------------------------- */

void __global__ vanity_init(unsigned long long int* rseed, hiprandState* state) {
	int id = threadIdx.x + (blockIdx.x * blockDim.x);  
	hiprand_init(*rseed + id, id, 0, &state[id]);
}

void __global__ vanity_scan(hiprandState* state, int* keys_found, int* gpu, int* exec_count) {
	int id = threadIdx.x + (blockIdx.x * blockDim.x);

        atomicAdd(exec_count, 1);

	// Local Kernel State
	ge_p3 A;
	hiprandState localState     = state[id];
	unsigned char seed[32]     = {0};
	unsigned char publick[32]  = {0};
	unsigned char privatek[64] = {0};
	char key[256]              = {0};
	//char pkey[256]             = {0};

	// Start from an Initial Random Seed (Slow)
	// NOTE: Insecure random number generator, do not use keys generator by
	// this program in live.
	// SMITH: localState should be entropy random now
	for (int i = 0; i < 32; ++i) {
		float random    = hiprand_uniform(&localState);
		uint8_t keybyte = (uint8_t)(random * 255);
		seed[i]         = keybyte;
	}

	// Generate Random Key Data
	sha512_context md;

	// I've unrolled all the MD5 calls and special cased them to 32 byte
	// inputs, which eliminates a lot of branching. This is a pretty poor
	// way to optimize GPU code though.
	//
	// A better approach would be to split this application into two
	// different kernels, one that is warp-efficient for SHA512 generation,
	// and another that is warp efficient for bignum division to more
	// efficiently scan for prefixes. Right now bs58enc cuts performance
	// from 16M keys on my machine per second to 4M.
	for (int attempts = 0; attempts < ATTEMPTS_PER_EXECUTION; ++attempts) {
		// sha512_init Inlined
		md.curlen   = 0;
		md.length   = 0;
		md.state[0] = UINT64_C(0x6a09e667f3bcc908);
		md.state[1] = UINT64_C(0xbb67ae8584caa73b);
		md.state[2] = UINT64_C(0x3c6ef372fe94f82b);
		md.state[3] = UINT64_C(0xa54ff53a5f1d36f1);
		md.state[4] = UINT64_C(0x510e527fade682d1);
		md.state[5] = UINT64_C(0x9b05688c2b3e6c1f);
		md.state[6] = UINT64_C(0x1f83d9abfb41bd6b);
		md.state[7] = UINT64_C(0x5be0cd19137e2179);

		// sha512_update inlined
		// 
		// All `if` statements from this function are eliminated if we
		// will only ever hash a 32 byte seed input. So inlining this
		// has a drastic speed improvement on GPUs.
		//
		// This means:
		//   * Normally we iterate for each 128 bytes of input, but we are always < 128. So no iteration.
		//   * We can eliminate a MIN(inlen, (128 - md.curlen)) comparison, specialize to 32, branch prediction improvement.
		//   * We can eliminate the in/inlen tracking as we will never subtract while under 128
		//   * As a result, the only thing update does is copy the bytes into the buffer.
		const unsigned char *in = seed;
		for (size_t i = 0; i < 32; i++) {
			md.buf[i + md.curlen] = in[i];
		}
		md.curlen += 32;


		// sha512_final inlined
		// 
		// As update was effectively elimiated, the only time we do
		// sha512_compress now is in the finalize function. We can also
		// optimize this:
		//
		// This means:
		//   * We don't need to care about the curlen > 112 check. Eliminating a branch.
		//   * We only need to run one round of sha512_compress, so we can inline it entirely as we don't need to unroll.
		md.length += md.curlen * UINT64_C(8);
		md.buf[md.curlen++] = (unsigned char)0x80;

		while (md.curlen < 120) {
			md.buf[md.curlen++] = (unsigned char)0;
		}

		STORE64H(md.length, md.buf+120);

		// Inline sha512_compress
		uint64_t S[8], W[80], t0, t1;
		int i;

		/* Copy state into S */
		for (i = 0; i < 8; i++) {
			S[i] = md.state[i];
		}

		/* Copy the state into 1024-bits into W[0..15] */
		for (i = 0; i < 16; i++) {
			LOAD64H(W[i], md.buf + (8*i));
		}

		/* Fill W[16..79] */
		for (i = 16; i < 80; i++) {
			W[i] = Gamma1(W[i - 2]) + W[i - 7] + Gamma0(W[i - 15]) + W[i - 16];
		}

		/* Compress */
		#define RND(a,b,c,d,e,f,g,h,i) \
		t0 = h + Sigma1(e) + Ch(e, f, g) + K[i] + W[i]; \
		t1 = Sigma0(a) + Maj(a, b, c);\
		d += t0; \
		h  = t0 + t1;

		for (i = 0; i < 80; i += 8) {
			RND(S[0],S[1],S[2],S[3],S[4],S[5],S[6],S[7],i+0);
			RND(S[7],S[0],S[1],S[2],S[3],S[4],S[5],S[6],i+1);
			RND(S[6],S[7],S[0],S[1],S[2],S[3],S[4],S[5],i+2);
			RND(S[5],S[6],S[7],S[0],S[1],S[2],S[3],S[4],i+3);
			RND(S[4],S[5],S[6],S[7],S[0],S[1],S[2],S[3],i+4);
			RND(S[3],S[4],S[5],S[6],S[7],S[0],S[1],S[2],i+5);
			RND(S[2],S[3],S[4],S[5],S[6],S[7],S[0],S[1],i+6);
			RND(S[1],S[2],S[3],S[4],S[5],S[6],S[7],S[0],i+7);
		}

		#undef RND

		/* Feedback */
		for (i = 0; i < 8; i++) {
			md.state[i] = md.state[i] + S[i];
		}

		// We can now output our finalized bytes into the output buffer.
		for (i = 0; i < 8; i++) {
			STORE64H(md.state[i], privatek+(8*i));
		}

		// Code Until here runs at 87_000_000H/s.

		// ed25519 Hash Clamping
		privatek[0]  &= 248;
		privatek[31] &= 63;
		privatek[31] |= 64;

		// ed25519 curve multiplication to extract a public key.
		ge_scalarmult_base(&A, privatek);
		ge_p3_tobytes(publick, &A);

		// Code Until here runs at 87_000_000H/s still!

		size_t keysize = 256;
		b58enc(key, &keysize, publick, 32);

		// Code Until here runs at 22_000_000H/s. b58enc badly needs optimization.

		// We don't have access to strncmp/strlen here, I don't know
		// what the efficient way of doing this on a GPU is, so I'll
		// start with a dumb loop. There seem to be implementations out
		// there of bignunm division done in parallel as a CUDA kernel
		// so it might make sense to write a new parallel kernel to do
		// this.

		if(key[0]=='p'&&key[1]=='o'&&key[2]=='w'&&key[3] >= '1' && key[3] <= '9'&&key[4] >= '1' && key[4] <= '9'&&key[5] >= '1' && key[5] <= '9')
		{

				for(int i=6;i<10;i++)
				{
					bool found=false;
						if (key[i] >= '1' && key[i] <= '9')
						{
							found=true;
						}
						else
						{
							break;
						}

					if(found)
					{
						{
                                        atomicAdd(keys_found, 1);


                                        printf("GPU %d MATCH %s - ", *gpu, key);
                                        for(int n=0; n<sizeof(seed); n++) { 
						printf("%02x",(unsigned char)seed[n]); 
					}
					printf("\n");
					
                                        printf("[");
					for(int n=0; n<sizeof(seed); n++) { 
						printf("%d,",(unsigned char)seed[n]); 
					}
                                        for(int n=0; n<sizeof(publick); n++) {
					        if ( n+1==sizeof(publick) ) {	
							printf("%d",publick[n]);
						} else {
							printf("%d,",publick[n]);
						}
					}
                                        printf("]\n");

				}
					}
				}

		} 
			

		// Code Until here runs at 22_000_000H/s. So the above is fast enough.

		// Increment Seed.
		// NOTE: This is horrifically insecure. Please don't use these
		// keys on live. This increment is just so we don't have to
		// invoke the CUDA random number generator for each hash to
		// boost performance a little. Easy key generation, awful
		// security.
		for (int i = 0; i < 32; ++i) {
			if (seed[i] == 255) {
				seed[i]  = 0;
			} else {
				seed[i] += 1;
				break;
			}
		}
	}

	// Copy Random State so that future calls of this kernel/thread/block
	// don't repeat their sequences.
	state[id] = localState;
}

bool __device__ b58enc(
	char    *b58,
       	size_t  *b58sz,
       	uint8_t *data,
       	size_t  binsz
) {
	// Base58 Lookup Table
	const char b58digits_ordered[] = "123456789ABCDEFGHJKLMNPQRSTUVWXYZabcdefghijkmnopqrstuvwxyz";

	const uint8_t *bin = data;
	int carry;
	size_t i, j, high, zcount = 0;
	size_t size;
	
	while (zcount < binsz && !bin[zcount])
		++zcount;
	
	size = (binsz - zcount) * 138 / 100 + 1;
	uint8_t buf[256];
	memset(buf, 0, size);
	
	for (i = zcount, high = size - 1; i < binsz; ++i, high = j)
	{
		for (carry = bin[i], j = size - 1; (j > high) || carry; --j)
		{
			carry += 256 * buf[j];
			buf[j] = carry % 58;
			carry /= 58;
			if (!j) {
				// Otherwise j wraps to maxint which is > high
				break;
			}
		}
	}
	
	for (j = 0; j < size && !buf[j]; ++j);
	
	if (*b58sz <= zcount + size - j) {
		*b58sz = zcount + size - j + 1;
		return false;
	}
	
	if (zcount) memset(b58, '1', zcount);
	for (i = zcount; j < size; ++i, ++j) b58[i] = b58digits_ordered[buf[j]];

	b58[i] = '\0';
	*b58sz = i + 1;
	
	return true;
}
